#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   make_fractols.c                                    :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: jlima-so <jlima-so@student.42lisba.com>    +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2025/06/16 16:51:48 by jlima-so          #+#    #+#             */
/*   Updated: 2025/06/21 13:46:20 by jlima-so         ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "../fractol.h"
#include "sys/time.h"

__host__ int 	fractol_formula(t_mlx_data *data)
{
	double	ind;
	double	real;

	ind = 0;
	while (ind++ < data->sc)
	{
		real = (data->n.z_r * data->n.z_r) - \
			(data->n.z_i * data->n.z_i) + data->n.c_r;
		data->n.z_i = (data->n.z_r * data->n.z_i * 2) + data->n.c_i;
		data->n.z_r = real;
		if ((data->n.z_r * data->n.z_r) + \
			(data->n.z_i * data->n.z_i) > 4)
		{
			if (data->col == XK_1 || data->col == XK_2 || data->col == XK_3)
				return ((((int)(((ind / data->sc)) * 255))) <<
				(8 * (data->col == XK_2) + 16 * (data->col == XK_3)));
			else if (data->col == XK_4)
				return ((ind / data->sc) * 16777216);
			else if (data->col == XK_5)
				return ((1 - (ind / data->sc)) * 16777216);
			return (data->a + data->b * cos(2 * 3.14159 \
				* (data->c * (ind / 100) + data->d)));
		}
	}
	return (0);
}

__device__ int fractol_formula_cuda(
    double z_r, double z_i, double c_r, double c_i,
    int sc, int col, double a, double b, double c, double d
) {
    double ind = 0;
    double real;
    while (ind++ < sc) {
        real = (z_r * z_r) - (z_i * z_i) + c_r;
        z_i = (z_r * z_i * 2) + c_i;
        z_r = real;
        if ((z_r * z_r) + (z_i * z_i) > 4) {
            if (col == XK_1 || col == XK_2 || col == XK_3)
                return ((((int)(((ind / sc)) * 255))) <<
                        (8 * (col == XK_2) + 16 * (col == XK_3)));
            else if (col == XK_4)
                return ((ind / sc) * 16777216);
            else if (col == XK_5)
                return ((1 - (ind / sc)) * 16777216);
            return (a + b * cos(2 * 3.14159 * (c * (ind / 100) + d)));
        }
    }
    return 0;
}

__global__ void julia_set(
    int *matrix, int width, int height,
    double x_mult, double y_mult,
    double x_cords, double y_cords,
    double real, double imag,
    int sc, int col, double a, double b, double c, double d
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        double z_r = ((x * x_mult / width)) - x_cords;
        double z_i = y_cords - ((y * y_mult) / height);
        double c_r = real;
        double c_i = imag;
        matrix[y * width + x] = fractol_formula_cuda(
            z_r, z_i, c_r, c_i, sc, col, a, b, c, d
        );
    }
}

__global__ void mandelbrot(
    int *matrix, int width, int height,
    double x_mult, double y_mult,
    double x_cords, double y_cords,
    int sc, int col, double a, double b, double c, double d
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        double z_r = 0.0;
        double z_i = 0.0;
        double c_r = ((x * x_mult / width)) - x_cords;
        double c_i = y_cords - ((y * y_mult) / height);
        matrix[y * width + x] = fractol_formula_cuda(
            z_r, z_i, c_r, c_i, sc, col, a, b, c, d
        );
    }
}

__host__ void show_frame(t_mlx_data *data) {
	int	x;
	int	y;
	int	w;
	int	h;

	y = -1;
	w = WIDTH;
	h = HIGHT;
	while (++y < h)
	{
		x = -1;
		while (++x < w)
		{
			my_put_pixel(&data->img, x, y, data->matrix[y * WIDTH + x]);
		}
	}
	aux_mlx_put_image_to_window(data->mlx_ptr, data->win_ptr, data->img.img_ptr, 0, 0);
}

__host__ void	j_m_bs(t_mlx_data *data)
{
	dim3 block(16, 16);
	dim3 grid((WIDTH + block.x - 1) / block.x, (HIGHT + block.y - 1) / block.y);

	// calculate with GPU
	if (data->flag == 1)
		julia_set<<<grid, block>>>(
			data->matrix, WIDTH, HIGHT,
			data->x_mult, data->y_mult,
			data->x_cords, data->y_cords,
			data->real, data->i,
			data->sc, data->col, data->a, data->b, data->c, data->d
		);
	else if (data->flag == 0)
		mandelbrot<<<grid, block>>>(
			data->matrix, WIDTH, HIGHT,
			data->x_mult, data->y_mult,
			data->x_cords, data->y_cords,
			data->sc, data->col, data->a, data->b, data->c, data->d
		);
	else
		burning_ship(data);


	// wait
	hipDeviceSynchronize();
	
	// show frame
	show_frame(data);

}

__host__ void create_matrix(int** matrix) {
	int count = 0;
	hipGetDeviceCount(&count);
	printf("CUDA device count: %d\n", count);
	if (count == 0) {
		fprintf(stderr, "No CUDA devices found!\n");
		exit(1);
	}
    hipMallocManaged(matrix, WIDTH * HIGHT * sizeof(int));
    int device = -1;
	hipError_t err = hipGetDevice(&device);
	printf("hipGetDevice returned %d, device = %d\n", err, device);
	hipMemPrefetchAsync(*matrix, WIDTH * HIGHT * sizeof(int), device, 0);
}

__host__ void delete_matrix(int* matrix) {
	hipFree(matrix);
}

/*int	second_julia_set(t_mlx_data *data, float r, float i)
{
	double	x;
	double	y;
	double	w;
	double	h;

	y = -1;
	w = WIDTH;
	h = HIGHT;
	while (++y < h)
	{
		x = -1;
		while (++x < w)
		{
			data->n.z_r = ((x * 2.65 / w)) - 1.325;
			data->n.z_i = 1.25 - ((y * 2.5) / h);
			data->n.c_r = r;
			data->n.c_i = i;
			my_put_pixel(&data->img2, x, y, fractol_formula(data));
		}
	}
	mlx_put_image_to_window
	(data->mlx_ptr, data->win_ptr2, data->img2.img_ptr, 0, 0);
	return (0);
}*/